#include "hip/hip_runtime.h"


/* This file is compiled with NVIDIA's compiler, nvcc. 
 *
 * nvcc enforces C++ semantics for linker symbols, which are 
 * different from those compiled as pure C files. Therefore
 * we need to declare all references to C compiled objects /
 * functions inside extern "C" brackets.                    */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <semaphore.h>
#include <time.h>
#include <pthread.h>
//#include <helper_functions.h>  // helper for shared functions common to CUDA Samples
//#include <hip/hip_runtime_api.h>       // helper functions for CUDA error checking and initialization
#include <mutex>
#include <queue>
#include <thread>
#include <iostream>
#include <fstream>

extern "C"{
#include "cudaforward.h"
#include "datatransfer-serial.h"
}

extern "C" {

    extern void** __cudaRegisterFatBinary(
            void *fatCubin
            );

    extern void __cudaUnregisterFatBinary(
            void **fatCubinHandle
            );

    extern void __cudaRegisterVar(
            void **fatCubinHandle,
            char *hostVar,
            char *deviceAddress,
            const char *deviceName,
            int ext,
            int size,
            int constant,
            int global
            );

    extern void __cudaRegisterTexture(
            void **fatCubinHandle,
            const struct textureReference *hostVar,
            const void **deviceAddress,
            const char *deviceName,
            int dim,
            int norm,
            int ext
            );

    extern void __cudaRegisterSurface(
            void **fatCubinHandle,
            const struct surfaceReference *hostVar,
            const void **deviceAddress,
            const char *deviceName,
            int dim,
            int ext
            );

    extern void __cudaRegisterFunction(
            void **fatCubinHandle,
            const char *hostFun,
            char *deviceFun,
            const char *deviceName,
            int thread_limit,
            uint3 *tid,
            uint3 *bid,
            dim3 *bDim,
            dim3 *gDim,
            int *wSize
            );
}

static std::mutex g_oMutex;
static std::queue<std::string> g_LogQueue;
static std::ofstream oFile("output.log");

static void OutputLog(const char* const szMessage);

void loggin_queue()
{
    while (true)
    {
        {
            std::lock_guard<std::mutex> lock(g_oMutex);
            while (g_LogQueue.size())
            {
                oFile << g_LogQueue.front() << std::endl;
                g_LogQueue.pop();
            }
        }
        //std::this_thread::sleep_for(10ms);
        usleep(100);
    }
}

/* Functions used to handle cudaRegisterFatBinary() / 
 * cudaUnregisterFatBinary() functions*/

/* Represents a fatbin file. Addociated with a guest PID. */
struct fatbinary{
    void* fatBin;
};

struct fatbinaryLL{
    struct fatbinary fatBinData;

    pid_t guest_pid;

    struct fatbinaryLL *next;
    struct fatbinaryLL *previous;
};

static struct fatbinaryLL *dummyFatBin;

static void addFatBinary(void *fatBin, pid_t guest_pid){

    struct fatbinaryLL *toAdd = (struct fatbinaryLL*) malloc(sizeof(struct fatbinaryLL));
    toAdd->fatBinData.fatBin = fatBin;
    toAdd->guest_pid = guest_pid;

    dummyFatBin->previous->next = toAdd;
    toAdd->previous = dummyFatBin->previous;
    toAdd->next = dummyFatBin;
    dummyFatBin->previous = toAdd;
}

static struct fatbinaryLL* getFatBinary(pid_t guest_pid){
    
    struct fatbinaryLL* search = dummyFatBin->next;
    while(search != dummyFatBin){
        if (search->guest_pid == guest_pid)
            return search;
        search = search->next;
    } 
    return NULL;
}

static void rmFatBinary(pid_t guest_pid){
    struct fatbinaryLL* search = getFatBinary(guest_pid);
    
    if(search == NULL)
        return;
    search->previous->next = search->next;
    search->next->previous = search->previous;
    free(search->fatBinData.fatBin);
    free(search);
}

/* Functions to use for cudaRegisterFunction() */

/* Represents a single kernel
 *
 * A guest cuda program can have many of these, depending on the number
 * of kernels in the binary. 
 *
 * NVCC uses a char pointer (not the string pointed to by it!) to identify
 * a kernel on launch, as opposed to using the kernel name directly from
 * a user-point-of-view. To correctly identify a kernel we thus need to be
 * able to map from this pointer to the (string) kernel name. */
struct kernLaunch{
    
    char* kernPtr;        // Pointer identifier
    char* kernName;       // Kernel name as in code
    void* fatBin;
};

// Linked list containing kernLaunch structures, identified by guest PID
struct kernLaunchLL{
    struct kernLaunch launchData;
    pid_t guest_pid;

    struct kernLaunchLL *next;
    struct kernLaunchLL *previous;
};

int fd; //shared memory location file descriptor
void* msg; //shared memory location base pointer
//pthread_spinlock_t *lock;

static struct kernLaunchLL *dummyKernLaunch;

int addKernFunc(const char* name, const char *kernPtr, pid_t guest_pid){
    
    struct kernLaunchLL *toAdd = 
        (struct kernLaunchLL*) malloc(sizeof(struct kernLaunchLL));

    struct fatbinaryLL *fatbinll = getFatBinary(guest_pid);
    if (fatbinll == NULL)
        return -1;

    struct fatbinary *fatbin = &fatbinll->fatBinData;

    toAdd->launchData.fatBin = fatbin->fatBin;
    toAdd->launchData.kernPtr = (char*)kernPtr;
    toAdd->launchData.kernName = (char*) malloc(strlen(name) + 1);
    strcpy(toAdd->launchData.kernName, name);
    toAdd->guest_pid = guest_pid;

    dummyKernLaunch->previous->next = toAdd;
    toAdd->previous = dummyKernLaunch->previous;
    dummyKernLaunch->previous = toAdd;
    toAdd->next = dummyKernLaunch;

    return 0;

}

struct kernLaunchLL* getKernFunc(void* name, pid_t guest_pid){
    
    struct kernLaunchLL *search = dummyKernLaunch->next;

    while(search != dummyKernLaunch){
        if(guest_pid == search->guest_pid && (void*)search->launchData.kernPtr == (void*)name)
            return search;
  
       search = search->next;
    }

    return NULL;
}


/* Removes all kernel functions associated with the given guest pid */
void rmKernFunc(pid_t guest_pid){

    struct kernLaunchLL* search = dummyKernLaunch->next;
    while(search != dummyKernLaunch){
        if(search->guest_pid == guest_pid){
            search->previous->next = search->next;
            search->next->previous = search->previous;
            free((void*) search->launchData.kernName);
            free((void*) search);
        }

        search = search->next;
    }

}

/* Functions to handle kernel configuration and launch:
 *
 * - hipConfigureCall()
 * - hipSetupArgument()
 * - hipLaunchByPtr() 
 *
 * These funcions build up a kernLaunchConfig data structure, which
 * is finally used for the hipLaunchByPtr call. */

struct kernLaunchArg{
    void* arg;

    struct kernLaunchArg *next;
    
};

struct kernLaunchConfig{

    dim3 gridDims;
    dim3 blockDims;

    size_t sh_mem;
    hipStream_t stream;

    unsigned int numArgs;
    struct kernLaunchArg *startArgs;

    pid_t guest_pid;

    struct kernLaunchConfig *next;
    struct kernLaunchConfig *previous;
    
};

static struct kernLaunchConfig* dummyLaunchConf;

struct kernLaunchConfig* getLaunchConfigStruct(pid_t guest_pid){
    
    struct kernLaunchConfig* search = dummyLaunchConf->next;

    while(search != dummyLaunchConf){
        if(search->guest_pid == guest_pid)
            return search;
        search = search->next;
    }
    return NULL;
}

int registerLaunchConfig(dim3 gridDims, dim3 blockDims, size_t sh_mem, hipStream_t stream, pid_t guest_pid){
   
    if(getLaunchConfigStruct(guest_pid) != NULL)
        return -1;

    struct kernLaunchConfig *toAdd = 
        (struct kernLaunchConfig*) malloc(sizeof(struct kernLaunchConfig));

    toAdd->gridDims = gridDims;
    toAdd->blockDims = blockDims;
    toAdd->sh_mem = sh_mem;
    toAdd->stream = stream;
    toAdd->guest_pid = guest_pid;
    toAdd->numArgs = 0;
    toAdd->startArgs = NULL;

    dummyLaunchConf->previous->next = toAdd;
    toAdd->previous = dummyLaunchConf->previous;
    toAdd->next = dummyLaunchConf;
    dummyLaunchConf->previous = toAdd;

    return 0;
}

int addLaunchParameter(pid_t guest_pid, const void* arg, size_t size, size_t offset){
    
    struct kernLaunchConfig* search = getLaunchConfigStruct(guest_pid);
    if(search != NULL){

        struct kernLaunchArg** searchArg = &search->startArgs;
        while(*searchArg != (void*) NULL){
            searchArg = &((*searchArg)->next);
        }
        *searchArg = 
            (struct kernLaunchArg*) malloc(sizeof(struct kernLaunchArg));
        struct kernLaunchArg *newLaunchParam = *searchArg;
        newLaunchParam->next = NULL;
        newLaunchParam->arg = malloc(size);
        memcpy(newLaunchParam->arg, arg, size);

        search->numArgs++;

        return 0;
        
    }
    return -1;
}

int removeLaunchConfig(pid_t guest_pid){
    
    struct kernLaunchConfig* search = getLaunchConfigStruct(guest_pid);
    
    if(search != NULL){
        struct kernLaunchArg* searchArg = search->startArgs;
        while(searchArg != NULL){
            void* toFree = (void*) searchArg;
            free(searchArg->arg);
            searchArg = searchArg->next;
            free(toFree);
        }

        search->previous->next = search->next;
        search->next->previous = search->previous;
        free((void*) search);

        return 0;
    }
    else{
        return -1;
    }

}



static void cudaFwdMsgHandler(uint32_t msg_sz, uint32_t *resMsgSz);

hipCtx_t context;

static int initContext(){

    //fprintf(stderr, "Context was reinitialized!\n");
    hipDevice_t device;
    if(hipDeviceGet(&device, 0) != hipSuccess){
        //fprintf(stderr, "Unable to get device 0! (Do you have a CUDA GPU?)\n");
        return -1;
    }
    if(hipCtxCreate(&context, hipDeviceScheduleAuto, device) != hipSuccess){
        fprintf(stderr, "Unable to create a context in cudaForward init!\n");
        return -1;
    }
    if(hipCtxPushCurrent(context) != hipSuccess){
        fprintf(stderr, "Unable to push the new context onto the current thread!\n");
        return -1;
    }


    return 0;


}

sem_t sem_in, sem_out;
//void *sem_bit;
clock_t begin,end;
double time_spent;
int *array;


struct cudaThread_inArgs{
    uint32_t msgSz;
};

struct cudaThread_outArgs{
    uint32_t respMsgSz;
};

struct cudaThread_outArgs *outArgs = (struct cudaThread_outArgs*) malloc(sizeof(struct cudaThread_outArgs));
struct cudaThread_inArgs *inArgs   = (struct cudaThread_inArgs*)  malloc(sizeof(struct cudaThread_inArgs));

/* Main CUDA thread. All cuda calls are executed here. This is to avoid having
 * to delve into context management which is internally handled with the CUDA
 * runtime API. 
 *
 * Ideally, one such thread should represent a thread in the guest. For our
 * purposes, it suffices having one. 
 *
 * Semaphores are posted by the cudaforward message handler at the bottom of
 * the file to wake up this thread. Subsequentially, this thread wakes up 
 * the handler function to handle the callback before going to sleep again. 
 * There is no need for a lock on the in/out argument data structures, as 
 * they are only accessed one at a time. This is dirty..  */
static void* mainCudaThread(void* threadArgs){

    // Need to do some trickery here to keep the script functions happy..
    void *respMsg_;
    void **respMsg = &respMsg_;

    uint32_t respMsgSz_;
    uint32_t *respMsgSz = &respMsgSz_;

    int status=1;
    
    while(status){
        //read(fd, respMsg_, sizeof(int));
        //printf("out.\n");
        //timespec time1;
    //clock_gettime(CLOCK_REALTIME, &time1);
    //printf("%li : %li\n", time1.tv_sec,time1.tv_nsec);
        sem_wait(&sem_in);
        //begin=clock();
        //fflush(stdout);
        //pthread_spin_lock(lock);
        //double time_spent;
        //printf("check : %d\n", inArgs->msgSz);

        *respMsgSz = 0;

        //uint8_t *mem; uint i;
        struct header *recHdr = (struct header*) msg;


        if(recHdr->cmdType == normCall){

//      fprintf(stderr, "Got normCall\n");
        struct callHeader *recCallHdr = (struct callHeader*) msg;
        //goto facudaRuntimeGetVersion;
        //printf("Call ID: %d\n", recCallHdr->callID+2);
// This file contains all our (auto-generated) forwarding functions.
#include "../python/forwarding_functions.cpp"
        }
        else if(recHdr->cmdType == asyncCall){

        }
        else if(recHdr->cmdType == kernelLaunch){

        }
        else{
            //fprintf(stderr, "Unrecognized commandtype!\n");
        }
done:
    //fflush(stdout);
    outArgs->respMsgSz = *respMsgSz;
    //mem = (uint8_t*) msg;
    //for (i=0; i < outArgs->respMsgSz; i++)
     //   mem[i]=*((uint8_t*)*respMsg+i);
    //clock_t begin;
    //begin = clock();
    //printf("post time : %d\n", begin);
    sem_post(&sem_out);
    //end = clock();
    //time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    //printf("time : %f\n", time_spent);
    //pthread_spin_unlock(lock);
    //printf("unlocked. %d\n", *lock);
    //fflush(stdout);
    }
    return (void*) 0;
}

int cudaforward_init(void){

// Uncomment to get a list of data structure sizes at startup
//#include "../../python/structsize.c"
    if(!oFile.is_open())
        std::cout<<"ERROR: Log file not configured properly!"<<std::endl;
    else
        std::cout<<"INFO: Log file configured properly!"<<std::endl;
    std::thread oLoggingQueue(loggin_queue);
    oLoggingQueue.detach();
    OutputLog("** STARTUP **");
    OutputLog("testing");
    registerLogger(OutputLog);
    sem_init(&sem_in, 0, 0);
    sem_init(&sem_out, 0, 0);
    
    if( access( "/dev/shm/mymem1", F_OK ) != -1 ) 
	{
		fprintf(stderr, "Opening mymem1 file!\n");
    	if ((fd=shm_open("mymem1", O_CREAT|O_RDWR|O_EXCL, S_IREAD | S_IWRITE)) > 0){} //shared memory stuff
    	else if ((fd=shm_open("mymem1", O_CREAT|O_RDWR, S_IREAD | S_IWRITE)) > 0){} 
    	else 
    	{
        	fprintf(stderr, "ERROR: cannot open file\n");
        	exit(-1);
    	}
    }
    else
    {
    	fprintf(stderr, "Opening mymem file!\n");
    	if ((fd=shm_open("mymem", O_CREAT|O_RDWR|O_EXCL, S_IREAD | S_IWRITE)) > 0){} //shared memory stuff
    	else if ((fd=shm_open("mymem", O_CREAT|O_RDWR, S_IREAD | S_IWRITE)) > 0){} 
    	else 
    	{
        	fprintf(stderr, "ERROR: cannot open file\n");
        	exit(-1);
    	}
    }
    
    
    if ((msg=mmap(NULL, 256*1024*1024, PROT_READ|PROT_WRITE, MAP_SHARED, fd, 0))<0){
        fprintf(stderr, "ERROR: cannot mmap file\n");
    }
    //sem_bit = ((uint8_t*)msg)+(255*1024*1024)-sizeof(pthread_spinlock_t);
    //lock = (pthread_spinlock_t*)sem_bit;
    //pthread_spin_init(lock,PTHREAD_PROCESS_SHARED);

    pthread_t cudaThread;
    pthread_create(&cudaThread, NULL, mainCudaThread, NULL);

    if(registerDataHandler(cudaFwdMsgHandler) != DTRANS_SUCCESS){
        
        fprintf(stderr, "Error registering the message handler!\n");
        return -1;
    }

    dummyLaunchConf = 
        (struct kernLaunchConfig*) malloc(sizeof(struct kernLaunchConfig));
    dummyFatBin = 
        (struct fatbinaryLL*) malloc(sizeof(struct fatbinaryLL));
    dummyKernLaunch = 
        (struct kernLaunchLL*) malloc(sizeof(struct kernLaunchLL));
    
    dummyLaunchConf->next = dummyLaunchConf;
    dummyLaunchConf->previous = dummyLaunchConf;

    dummyFatBin->next = dummyFatBin;
    dummyFatBin->previous = dummyFatBin;

    dummyKernLaunch->next = dummyKernLaunch;
    dummyKernLaunch->previous = dummyKernLaunch;

    //fprintf(stderr, "\ndummyLaunchConf: %lx\ndummyFatBin: %lx\ndummyKernLaunch: %lx\n", 
    //        dummyLaunchConf,
    //        dummyFatBin,
    //        dummyKernLaunch);

    if(dummyLaunchConf == 0 || dummyFatBin == 0 || dummyKernLaunch == 0)
        return -1;

    if(hipInit(0) != hipSuccess)
        return -1;

    // This is a horrible way to initialize contexts. It should be done
    // on a per-process-in-guest basis; buf for now, all guest processes
    // actually share a context. 
    if(initContext() != 0)
        return -1;

    if(hipCtxPopCurrent(&context) != hipSuccess){
        fprintf(stderr, "Unable to pop the context in cudaforward init!\n");
        return -1;
    }
    return 0;
}


/* For the libvirt data transfers using datatransfer-serial.h, this handler function is
 * executed as its own thread. That means, as it is executed, it will not have a context.
 * This creates a problem in cases where the guest calls driver api functions. This is solved
 * by creating a main cuda handler thread. The CUDA API can the manage contexts itself, 
 * as is normal. Please see the comment in the main cuda handler function above. 
 *
 * We have to manually set and unset the context on enter and leave to this function.
 *
 * For now, it is garuanteed that only one handler exists at a time (to avoid race conditions
 * and a lot of work..) */
static void cudaFwdMsgHandler(uint32_t msg_sz, uint32_t *respMsgSz){

    inArgs->msgSz = msg_sz;
    sem_post(&sem_in);
    sem_wait(&sem_out);

    *respMsgSz = outArgs->respMsgSz;

}

static void OutputLog(const char *const szMessage)
{
    std::lock_guard<std::mutex> lock(g_oMutex);
    g_LogQueue.push(szMessage);
}

sem_t* getsemaphores(void){
    return &sem_in;
}
