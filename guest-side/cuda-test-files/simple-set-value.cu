
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void setValue(int* c)
{
    printf("DEVICE computation!\n");
    *c = 4;
}

int main()
{
    int c; // host copies of a,b,c
    int *d_c; // device copies of a,b,c
    int size = sizeof(int);

    // allocate space for device copies of a,b,c
    hipError_t error_id = hipMalloc((void**)&d_c,size);
    if (error_id != hipSuccess)
    {
        printf("hipMalloc returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
      	exit(EXIT_FAILURE);
    }

    // launch add() kernel on GPU
    setValue<<<1,1>>>(d_c);

    // copy result back to host
    hipMemcpy(&c,d_c,size,hipMemcpyDeviceToHost);

    printf("%d\n",c);
    
    hipFree(d_c);

    return 0;
}
