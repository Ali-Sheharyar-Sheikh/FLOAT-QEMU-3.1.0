
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add1(int* a, int* b, int* c)
{
    printf("Param a value: %d\n",*a);
    printf("Param b value: %d\n",*b);
    *c = *a + *b;
    printf("Result c value: %d\n",*c);
}

int main()
{
    int *a,*b,*c; // host copies of a,b,c
    int *d_a,*d_b,*d_c; // device copies of a,b,c
    int size = sizeof(int);

    // allocate space for device copies of a,b,c
    hipError_t error_id = hipMalloc((void**)&d_a,size);
    if (error_id != hipSuccess)
    {
        printf("hipMalloc returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }
    hipMalloc((void**)&d_b,size);
    hipMalloc((void**)&d_c,size);

    // setup input variables
    a = (int*)malloc(sizeof(int));
    b = (int*)malloc(sizeof(int));
    c = (int*)malloc(sizeof(int));
    
    for(int i=1;i<500;i++)
    {
   
   	*a=i;
   	*b=300;

    // copy inputs to device
    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
    //cudaMemcpy(d_c,c,size,cudaMemcpyHostToDevice);

    // launch add1() kernel on GPU
    add1<<<1,1>>>(d_a,d_b,d_c);

    // copy result back to host
    hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	
	fprintf(stderr,"HOST computation!\n");
    fprintf(stderr,"A: %d + B: %d\n",*a,*b);
	fprintf(stderr,"Result C: %d.\n",*c);
   
    }
    
	free(a);
	free(b);
	free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
