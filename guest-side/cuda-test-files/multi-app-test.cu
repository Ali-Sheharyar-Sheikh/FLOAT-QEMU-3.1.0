
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <thread>
#include <chrono>

__global__ void add2(int* a, int* b, int* c)
{
    printf("Param a value: %d\n",*a);
    printf("Param b value: %d\n",*b);
    *c = *a + *b;
    printf("Result c value: %d\n",*c);
}

int main(int argc, char** argv)
{
    int *a,*b,*c; // host copies of a,b,c
    int *d_a,*d_b,*d_c; // device copies of a,b,c
    int size = sizeof(int);
	int nKernelMaxLimit = 50;
	if(argc>1)
		nKernelMaxLimit = atoi(argv[1]);
	
    // allocate space for device copies of a,b,c
    hipError_t error_id = hipMalloc((void**)&d_a,size);
    if (error_id != hipSuccess)
    {
        printf("hipMalloc returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }
    hipMalloc((void**)&d_b,size);
    hipMalloc((void**)&d_c,size);

    // setup input variables
    a = (int*)malloc(sizeof(int));
    b = (int*)malloc(sizeof(int));
    c = (int*)malloc(sizeof(int));
    
    for(int i=1;i<nKernelMaxLimit;i++)
    {
   
   	*a=i;
   	*b=5;

    // copy inputs to device
    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
    //cudaMemcpy(d_c,c,size,cudaMemcpyHostToDevice);

    // launch add1() kernel on GPU
    add2<<<1,1>>>(d_a,d_b,d_c);

    // copy result back to host
    hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	
	fprintf(stderr,"HOST computation!\n");
    fprintf(stderr,"A: %d + B: %d\n",*a,*b);
	fprintf(stderr,"Result C: %d.\n",*c);
   
   	std::this_thread::sleep_for(std::chrono::milliseconds(250));
   
    }
    
	free(a);
	free(b);
	free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}