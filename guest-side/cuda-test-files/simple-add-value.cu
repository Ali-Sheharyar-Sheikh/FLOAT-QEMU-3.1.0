
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int* a, int* b, int* c)
{
    *c = *a + *b;
	printf("simple add value\n");
}

int main()
{
    int a,b,c; // host copies of a,b,c
    int *d_a,*d_b,*d_c; // device copies of a,b,c
    int size = sizeof(int);

    // allocate space for device copies of a,b,c
    hipError_t error_id = hipMalloc((void**)&d_a,size);
    if (error_id != hipSuccess)
    {
        printf("hipMalloc returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }
    hipMalloc((void**)&d_b,size);
    hipMalloc((void**)&d_c,size);

    // setup input variables
    a=2;
    b=7;

    // copy inputs to device
    hipMemcpy(d_a,&a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,&b,size,hipMemcpyHostToDevice);
    hipMemcpy(d_c,&c,size,hipMemcpyHostToDevice);

    // launch add() kernel on GPU
    add<<<1,1>>>(d_a,d_b,d_c);

    // copy result back to host
    hipMemcpy(&c,d_c,size,hipMemcpyDeviceToHost);

    printf("%d\n",c);
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
