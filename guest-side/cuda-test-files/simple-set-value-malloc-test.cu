
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void setValue(int* c)
{
    printf("DEVICE computation!\n");
    *c = 18;
    printf("Result C: %d!\n",*c);
    printf("----\n");
}

int main()
{ 
    printf("** Simple Set Value Malloc Test **\n");
    int *c; // host copies of a,b,c
    int *d_c; // device copies of a,b,c
    int size = sizeof(int);

    // allocate space for device copies of a,b,c
    hipError_t error_id = hipMalloc((void**)&d_c,size);
    if (error_id != hipSuccess)
    {
        printf("hipMalloc returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
      	exit(EXIT_FAILURE);
    }

    // launch add() kernel on GPU
    setValue<<<1,1>>>(d_c);

    c = (int*)malloc(sizeof(int));

    // copy result back to host
    hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
    
    printf("Host Computation!\n");

    printf("Result C: %d\n",*c);
    printf("----\n");
    
    hipFree(d_c);

    return 0;
}
