
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void setValue(int* c)
{
    printf("DEVICE computation!\n");
    *c = 25;
    printf("Result C: %d!\n",*c);
    printf("----\n");
}

int main()
{ 
    int *c; // host copies of a,b,c
    int *d_c; // device copies of a,b,c
    int size = sizeof(int);

    // allocate space for device copies of a,b,c
    hipError_t error_id = hipMalloc((void**)&d_c,size);
    if (error_id != hipSuccess)
    {
        printf("hipMalloc returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
      	exit(EXIT_FAILURE);
    }

    // launch add() kernel on GPU
    setValue<<<1,1>>>(d_c);

    c = (int*)malloc(sizeof(int));

    // copy result back to host
    hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
    
    fprintf(stderr,"Result C: %d\n",*c);
    
	free(c);
    hipFree(d_c);

    return 0;
}
