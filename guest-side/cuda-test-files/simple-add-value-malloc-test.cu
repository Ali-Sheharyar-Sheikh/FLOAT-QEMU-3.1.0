
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int* a, int* b, int* c)
{
    printf("Param a value: %d\n",*a);
    printf("Param b value: %d\n",*b);
    *c = *a + *b;
    printf("Result c value: %d\n",*c);
}

int main()
{
    int *a,*b,*c; // host copies of a,b,c
    int *d_a,*d_b,*d_c; // device copies of a,b,c
    int size = sizeof(int);

    // allocate space for device copies of a,b,c
    hipError_t error_id = hipMalloc((void**)&d_a,size);
    if (error_id != hipSuccess)
    {
        printf("hipMalloc returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }
    hipMalloc((void**)&d_b,size);
    hipMalloc((void**)&d_c,size);

    // setup input variables
    a = (int*)malloc(sizeof(int));
    b = (int*)malloc(sizeof(int));
    c = (int*)malloc(sizeof(int));
    *a=300;
    *b=117;

    // copy inputs to device
    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
    hipMemcpy(d_c,c,size,hipMemcpyHostToDevice);

    // launch add() kernel on GPU
    add<<<1,1>>>(d_a,d_b,d_c);

    // copy result back to host
    hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	
	fprintf(stderr,"HOST computation!\n");
    fprintf(stderr,"A: %d + B: %d\n",*a,*b);
	fprintf(stderr,"Result C: %d.\n",*c);
    
	free(a);
	free(b);
	free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
